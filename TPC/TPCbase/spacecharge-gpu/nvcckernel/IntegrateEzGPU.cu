#include "hip/hip_runtime.h"
#include "IntegrateEzGPU.h"

#include <hip/hip_runtime.h>
#include <math.h>

__device__ __constant__ float d_gridSizeZ;
__device__ __constant__ float d_ezField;
__device__ __constant__ int d_scanSize;
__device__ __constant__ int d_nRRow;
__device__ __constant__ int d_nZColumn;
__device__ __constant__ int d_phiSlice;
__device__ __constant__ int d_interpolationOrder;
__device__ int d_currentZIndex;


__global__ void integrationCalculation
(
	float *d_arrayofIntEx,
	float *d_arrayofEx	
)
{
	extern __shared__ float temp[];

	int threadIndex = threadIdx.x;	
	int arrayIndex = blockIdx.x * (d_scanSize + 1);
	
	float first, second, last;
	
	int n = blockDim.x * 2;

	int offset = 1;
	
	// load data from input
	float temp_a = d_arrayofEx[arrayIndex + (2 * threadIndex)];
	float temp_b = d_arrayofEx[arrayIndex + (2 * threadIndex + 1)];

	// load last element from array to first variable
	first = d_arrayofEx[arrayIndex + d_scanSize];
	second = d_arrayofEx[arrayIndex + d_scanSize - 1];

/* odd function */
	// save data to shared memory flipped
	temp[(d_scanSize - 1) - (2 * threadIndex)] = 4 * temp_a;
	temp[(d_scanSize - 1) - (2 * threadIndex + 1)] = 2 * temp_b;
	
	// scan the array
	for (int d = n >> 1; d > 0; d >>= 1)
	{
		__syncthreads();
		
		if (threadIndex < d)
		{
			int ai = offset * (2 * threadIndex + 1) - 1;
			int bi = offset * (2 * threadIndex + 2) - 1;

			temp[bi] += temp[ai];
		}
		offset *= 2;
	}

	if (threadIndex == 0)
	{
		temp[n - 1] = 0;
	}

	for (int d = 1; d < n; d *= 2)
	{
		offset >>= 1;
		__syncthreads();

		if (threadIndex < d)
		{
			int ai = offset * (2 * threadIndex + 1) - 1;
			int bi = offset * (2 * threadIndex + 2) - 1;

			float t = temp[ai];
			temp[ai] = temp[bi];
			temp[bi] += t;
		}
	}
	__syncthreads();
	
	// save odd-numbered scan to even-numbered array
	d_arrayofIntEx[arrayIndex + (2 * threadIndex + 1)] = ((1.5 * first) + (0.5 * second) + temp[(d_scanSize - 1) - (2 * threadIndex)] - temp_b) * (d_gridSizeZ / 3.0) / (-1 * d_ezField);

/* even function */
	// save data to shared memory flipped
	temp[(d_scanSize - 1) - (2 * threadIndex)] = 2 * temp_a;
	temp[(d_scanSize - 1) - (2 * threadIndex + 1)] = 4 * temp_b;
	
	// scan the array
	for (int d = n >> 1; d > 0; d >>= 1)
	{
		__syncthreads();
		
		if (threadIndex < d)
		{
			int ai = offset * (2 * threadIndex + 1) - 1;
			int bi = offset * (2 * threadIndex + 2) - 1;

			temp[bi] += temp[ai];
		}
		offset *= 2;
	}

	if (threadIndex == 0)
	{
		last = temp[n - 1];		
		temp[n - 1] = 0;
	}

	for (int d = 1; d < n; d *= 2)
	{
		offset >>= 1;
		__syncthreads();

		if (threadIndex < d)
		{
			int ai = offset * (2 * threadIndex + 1) - 1;
			int bi = offset * (2 * threadIndex + 2) - 1;

			float t = temp[ai];
			temp[ai] = temp[bi];
			temp[bi] += t;
		}
	}
	__syncthreads();

	if (threadIndex == 0)
	{
		d_arrayofIntEx[arrayIndex + d_scanSize] = 0.0;
		d_arrayofIntEx[arrayIndex] = (first + last - temp_a) * (d_gridSizeZ / 3.0) / (-1 * d_ezField);
	}
	else
	{
		d_arrayofIntEx[arrayIndex + (2 * threadIndex)] = (first + temp[(d_scanSize - 1) - (2 * threadIndex) + 1] - temp_a) * (d_gridSizeZ / 3.0) / (-1 * d_ezField);
	}
}

extern "C" void IntegrateEzGPU 
(
	float *arrayOfIntEx, 
	float *arrayOfEx, 
	const int rows, 
	const int columns,  
	const int phislices, 
	float gridSizeZ, 
	float ezField	
)
{
	// initialize device array
	float *d_arrayofIntEx;
	float *d_arrayofEx;

	// set scan size to columns - 1
	int scanSize = columns - 1;

	std::cout << scanSize << std::endl;

	// set grid size and block size
	dim3 gridSize(rows * phislices);
	dim3 blockSize(scanSize / 2);

	// device memory allocation
	hipMalloc( &d_arrayofIntEx, rows * columns * phislices * sizeof(float) );
	hipMalloc( &d_arrayofEx, rows * columns * phislices * sizeof(float) );

	// copy data from host to device
	hipMemcpy( d_arrayofEx, arrayOfEx, rows * columns * phislices * sizeof(float), hipMemcpyHostToDevice );

	// copy constant to device memory
	hipMemcpyToSymbol(HIP_SYMBOL( d_gridSizeZ), &gridSizeZ, 1 * sizeof(float), 0, hipMemcpyHostToDevice );
	hipMemcpyToSymbol(HIP_SYMBOL( d_ezField), &ezField, 1 * sizeof(float), 0, hipMemcpyHostToDevice );
	hipMemcpyToSymbol(HIP_SYMBOL( d_scanSize), &scanSize, 1 * sizeof(int), 0, hipMemcpyHostToDevice );

	// run the kernel
	integrationCalculation<<< gridSize, blockSize, 2 * scanSize * sizeof(float) >>>( d_arrayofIntEx, d_arrayofEx );

	// copy result from device to host
	hipMemcpy( arrayOfIntEx, d_arrayofIntEx, rows * columns * phislices * sizeof(float), hipMemcpyDeviceToHost );

	// free device memory
	hipFree( d_arrayofIntEx );
	hipFree( d_arrayofEx );
}

__device__ void interpolatePhiGPU
(
	float * xList,
	int iLow,
	int lenX,
	float * yList,
	float x,
	float *y
)
{
	
  	int i0 = iLow;
  	float xi0 = xList[iLow];
  	int  i1 = (iLow + 1) % lenX;
  	float xi1 = xList[i1];
  	int  i2 = (iLow + 2) % lenX;
  	float  xi2 = xList[i2];

  	if (xi1 < xi0) xi1 = 2*M_PI + xi1;
  	if (xi2 < xi1) xi2 = 2*M_PI + xi2;
  	if (x < xi0) x = 2*M_PI + x;

    	*y = yList[0] + (yList[1] - yList[0]) * (x - xi0) / (xi1 - xi0);
}

__device__ void interpolateGPU
(
	float * xList,
	float * yList,
	float x,
	float * y
)
{
	
    	*y = yList[0] + (yList[1] - yList[0]) * (x - xList[0]) / (xList[1] - xList[0]);
}

__device__ void interpolate3DCylindricalGPU
(	
	int iLow,
	int jLow,
	int kLow,
	float r,
	float z,
	float phi,
	float *rList,
	float *zList,
	float *phiList,
	float *valList,
	float *interpolationValue
)
{

  	// do for each
	int m,index;
	float saveArray[5];
	float savedArray[5];
	float zListM1[3];
	float valueM1[3];

  	for (int k = 0; k < d_interpolationOrder + 1; k++) {
    		m = (kLow + k) % d_phiSlice;
    		// interpolate
    		for (int i = iLow; i < iLow + d_interpolationOrder + 1; i++) {
      			if (d_interpolationOrder <= 2) {
        			if (jLow >= 0) {
          				index = m * (d_nZColumn  * d_nRRow) + i * (d_nZColumn) + jLow;
          				interpolateGPU(&zList[jLow], &valList[index], z,&saveArray[i - iLow]);
        			} else {
					
          				index = m * (d_nZColumn* d_nRRow) + i * (d_nZColumn);
          				zListM1[0] = zList[0] - (zList[1] - zList[0]);
          				zListM1[1] = zList[0];
          				zListM1[2] = zList[1];
          				valueM1[0] = valList[index] - (valList[index + 1] - valList[index]);
          				valueM1[1] = valList[index];
          				valueM1[2] = valList[index + 1];
         				interpolateGPU(&zListM1[0], &valueM1[0], z,&saveArray[i-iLow]);
        			}

      			}
    		}
    		interpolateGPU(&rList[iLow], saveArray, r,&savedArray[k]);
  	}
  	interpolatePhiGPU(phiList, kLow, d_phiSlice, savedArray, phi,interpolationValue);
}
		
			



__global__ void integrateDistEzDriftLineGPUKernel
(
	float *distDrDz,
	float *distDPhiRDz,
	float *distDz, 
	float *GDistDrDz,
	float *GDistDPhiRDz,
	float *GDistDz, 
	float *rList,
	float *zList,
	float *phiList,
	float *secondDerZDistDr,
	float *secondDerZDistDPhiR,
	float *secondDerZDistDz,
	int operationType,
	int currentZIndex
)
{
	int index, index_x, index_y, index_z;
	
	// float gDistDrDz, gDistDPhiRDz, gDistDz;
	float lDistDrDz, lDistDPhiRDz, lDistDz;
	float currentPhi,currentRadius, currentZ;
	
	// for interpolation
	int iLow,jLow,kLow;

	
	index = (blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

	index_x = index / (d_nZColumn * d_nRRow);
	
	if (index_x == 0)
	{
		index_y = index / d_nRRow;
	}
	else	
	{
		index_y = (index % (index_x * d_nRRow * d_nZColumn)) / d_nRRow;
	}

	index_z = index % d_nZColumn;
	


	index = index_x * d_nRRow * d_nZColumn + index_y * d_nZColumn + index_z;	

	if (operationType == 0) {
	   if ((index_x >= 0) && (index_x < d_phiSlice) && (index_y >= 0) && (index_y < d_nRRow ) && (index_z >= 0) && (index_z < d_nZColumn)) 			{
			
		GDistDrDz[index] = 0.0;
		GDistDPhiRDz[index] = 0.0;
		GDistDz[index] = 0.0;
		}
	} else {
	   if ((index_x >= 0) && (index_x < d_phiSlice) && (index_y >= 0) && (index_y < d_nRRow ) && (index_z >= 0) && (index_z < (d_nZColumn - 1)) && (index_z <=  currentZIndex)) {
		lDistDrDz = 0.0;
		lDistDPhiRDz = 0.0;
		lDistDz = 0.0;
		
		currentRadius = rList[index_y] + GDistDrDz[index];
		currentPhi = phiList[index_x] + (GDistDPhiRDz[index]/currentRadius);
		if (currentPhi < 0.0) currentPhi = 2 * M_PI + currentPhi;
		if (currentPhi > 2*M_PI) currentPhi = currentPhi - (2 * M_PI);
		currentZ =  zList[currentZIndex] + GDistDz[index];

		// get Local Distortion through interpolation
		// get nearest index
		for (iLow=0;iLow<d_nRRow;iLow++)  if (rList[iLow] > currentRadius) break;
		for (jLow=0;jLow<d_nZColumn;jLow++)  if (zList[jLow] > currentZ) break;
		for (kLow=0;kLow<d_phiSlice;kLow++)  if (phiList[kLow] > currentPhi) break;

		iLow--;jLow--;kLow--;
  		// order >= 3
  		kLow -= (d_interpolationOrder / 2);
  		iLow -= (d_interpolationOrder / 2);
  		jLow -= (d_interpolationOrder / 2);

  		// check if out of range
  		if (iLow < 0) iLow = 0;
  		if (jLow < 0) jLow = 0;
  		if (kLow < 0) kLow = d_phiSlice + kLow;
  		// check if out of range
 		if (iLow + d_interpolationOrder >= d_nRRow  - 1) iLow = d_nRRow- 1 - d_interpolationOrder;
  		if (jLow + d_interpolationOrder >= d_nZColumn - 1) jLow = d_nZColumn - 1 - d_interpolationOrder;

		interpolate3DCylindricalGPU(iLow,jLow,kLow,currentRadius,currentZ,currentPhi,rList,zList,phiList,distDrDz,&lDistDrDz);	
		interpolate3DCylindricalGPU(iLow,jLow,kLow,currentRadius,currentZ,currentPhi,rList,zList,phiList,distDPhiRDz,&lDistDPhiRDz);	
		interpolate3DCylindricalGPU(iLow,jLow,kLow,currentRadius,currentZ,currentPhi,rList,zList,phiList,distDz,&lDistDz);	
			
		// update global distortion
		GDistDrDz[index] += lDistDrDz;
		GDistDPhiRDz[index] += lDistDPhiRDz;
		GDistDz[index] += lDistDz;
		
			
	    }
	}

}



extern "C" void IntegrateEzDriftLineGPU(
	float * distDrDz, float * distDPhiRDz, float * distDz, float *corrDrDz, float * corrDPhiRDz, float * corrDz,  
	float * GDistDrDz, float * GDistDPhiRDz, float * GDistDz, float * GCorrDrDz, float * GCorrDPhiRDz, float * GCorrDz,  
	float * rList, float * zList, float * phiList,   
	const int rows, const int columns, const int phislices, const int interpolationOrder,
	float * secondDerZDistDr, float *secondDerZDistDPhiR, float *secondDerZDistDz,
	float * secondDerZCorrDr, float *secondDerZCorrDPhiR, float *secondDerZCorrDz) {



	// initialize device array
	float *d_distDrDz;
	float *d_distDPhiRDz;
	float *d_distDz;
	float *d_corrDrDz;
	float *d_corrDPhiRDz;
	float *d_corrDz;
	float *d_GDistDrDz;
	float *d_GDistDPhiRDz;
	float *d_GDistDz;
	float *d_GCorrDrDz;
	float *d_GCorrDPhiRDz;
	float *d_GCorrDz;
	float *d_rList;
	float *d_zList;
	float *d_phiList;
	float *d_secondDerZDistDr;
	float *d_secondDerZDistDPhiR;
	float *d_secondDerZDistDz;

	float *d_secondDerZCorrDr;
	float *d_secondDerZCorrDPhiR;
	float *d_secondDerZCorrDz;

	int currentZIndex;

	hipError_t error;

	hipMalloc( &d_distDrDz, rows * columns * phislices * sizeof(float) );
	hipMalloc( &d_distDPhiRDz, rows * columns * phislices * sizeof(float) );
	hipMalloc( &d_distDz, rows * columns * phislices * sizeof(float) );
	hipMalloc( &d_corrDrDz, rows * columns * phislices * sizeof(float) );
	hipMalloc( &d_corrDPhiRDz, rows * columns * phislices * sizeof(float) );
	hipMalloc( &d_corrDz, rows * columns * phislices * sizeof(float) );
	hipMalloc( &d_GDistDrDz, rows * columns * phislices * sizeof(float) );
	hipMalloc( &d_GDistDPhiRDz, rows * columns * phislices * sizeof(float) );
	hipMalloc( &d_GDistDz, rows * columns * phislices * sizeof(float) );
	hipMalloc( &d_GCorrDrDz, rows * columns * phislices * sizeof(float) );
	hipMalloc( &d_GCorrDPhiRDz, rows * columns * phislices * sizeof(float) );
	hipMalloc( &d_GCorrDz, rows * columns * phislices * sizeof(float) );
	hipMalloc( &d_rList, rows *  sizeof(float) );
	hipMalloc( &d_zList, columns *  sizeof(float) );
	hipMalloc( &d_phiList,  phislices * sizeof(float) );

	hipMalloc( &d_secondDerZDistDr, rows *  columns * phislices *  sizeof(float) );
	hipMalloc( &d_secondDerZDistDPhiR, rows *  columns * phislices *  sizeof(float) );
	hipMalloc( &d_secondDerZDistDz, rows *  columns * phislices *  sizeof(float) );
	
	hipMalloc( &d_secondDerZCorrDr, rows *  columns * phislices *  sizeof(float) );
	hipMalloc( &d_secondDerZCorrDPhiR, rows *  columns * phislices *  sizeof(float) );
	hipMalloc( &d_secondDerZCorrDz, rows *  columns * phislices *  sizeof(float) );



	error = hipGetLastError();	
	if ( error != hipSuccess )
	{    	
		std::cout << "CUDA memory allocation error: " << hipGetErrorString(error) << '\n';
	}


	// copy from CPU to GPU
	// copy local distortion 
	hipMemcpy( d_distDrDz, distDrDz, rows * columns * phislices * sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( d_distDPhiRDz, distDPhiRDz, rows * columns * phislices * sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( d_distDz, distDz, rows * columns * phislices * sizeof(float), hipMemcpyHostToDevice );

	hipMemcpy( d_corrDrDz, corrDrDz, rows * columns * phislices * sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( d_corrDPhiRDz, corrDPhiRDz, rows * columns * phislices * sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( d_corrDz, corrDz, rows * columns * phislices * sizeof(float), hipMemcpyHostToDevice );

	hipMemcpy( d_rList, rList, rows  * sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( d_zList, zList, columns *  sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( d_phiList, phiList,  phislices * sizeof(float), hipMemcpyHostToDevice );
	
	hipMemcpy( d_secondDerZDistDr, secondDerZDistDr, rows * columns * phislices * sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( d_secondDerZDistDPhiR, secondDerZDistDPhiR, rows * columns * phislices * sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( d_secondDerZDistDz, secondDerZDistDz, rows * columns * phislices * sizeof(float), hipMemcpyHostToDevice );

	hipMemcpy( d_secondDerZCorrDr, secondDerZCorrDr, rows * columns * phislices * sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( d_secondDerZCorrDPhiR, secondDerZCorrDPhiR, rows * columns * phislices * sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( d_secondDerZCorrDz, secondDerZCorrDz, rows * columns * phislices * sizeof(float), hipMemcpyHostToDevice );
	
	hipMemcpyToSymbol(HIP_SYMBOL( d_interpolationOrder), &interpolationOrder, 1 * sizeof(int), 0, hipMemcpyHostToDevice );
	hipMemcpyToSymbol(HIP_SYMBOL( d_nRRow), &rows, 1 * sizeof(int), 0, hipMemcpyHostToDevice );
	hipMemcpyToSymbol(HIP_SYMBOL( d_nZColumn), &columns, 1 * sizeof(int), 0, hipMemcpyHostToDevice );
	hipMemcpyToSymbol(HIP_SYMBOL( d_phiSlice), &phislices, 1 * sizeof(int), 0, hipMemcpyHostToDevice );
	error = hipGetLastError();	
	if ( error != hipSuccess )
	{
		std::cout << "CUDA memory copy host to device error: " << hipGetErrorString(error) << '\n';
	}

	// call kernel
	// set grid size and block size
	dim3 gridSize((rows / 32) + 1, (columns / 32) + 1, phislices);
	dim3 blockSize(32, 32);

	integrateDistEzDriftLineGPUKernel<<< gridSize,blockSize >>>(d_distDrDz,d_distDPhiRDz,d_distDz,
					  d_GDistDrDz,d_GDistDPhiRDz,d_GDistDz, 
					  d_rList,d_zList, d_phiList,
					  d_secondDerZDistDr, d_secondDerZDistDPhiR, d_secondDerZDistDz,0,currentZIndex);
	for (currentZIndex = 0; currentZIndex < columns;currentZIndex++) {	
		integrateDistEzDriftLineGPUKernel<<< gridSize,blockSize >>>(d_distDrDz,d_distDPhiRDz,d_distDz,
					  d_GDistDrDz,d_GDistDPhiRDz,d_GDistDz, 
					  d_rList,d_zList, d_phiList,
					  d_secondDerZDistDr, d_secondDerZDistDPhiR, d_secondDerZDistDz,1,currentZIndex);

	}
	
	hipMemcpy( GDistDrDz, d_GDistDrDz, rows * columns * phislices * sizeof(float), hipMemcpyDeviceToHost );
	hipMemcpy( GDistDPhiRDz, d_GDistDPhiRDz, rows * columns * phislices * sizeof(float), hipMemcpyDeviceToHost );
	hipMemcpy( GDistDz, d_GDistDz, rows * columns * phislices * sizeof(float), hipMemcpyDeviceToHost );
	hipMemcpy( GCorrDrDz, d_GCorrDrDz, rows * columns * phislices * sizeof(float), hipMemcpyDeviceToHost );
	hipMemcpy( GCorrDPhiRDz, d_GCorrDPhiRDz, rows * columns * phislices * sizeof(float), hipMemcpyDeviceToHost );
	hipMemcpy( GCorrDz, d_GCorrDz, rows * columns * phislices * sizeof(float), hipMemcpyDeviceToHost );
	error = hipGetLastError();	
	if ( error != hipSuccess )
	{
		std::cout << "CUDA Global dist  memory copy device to host error: " << hipGetErrorString(error) << '\n';
	}


	hipFree( d_distDrDz );
	hipFree( d_distDPhiRDz );
	hipFree( d_distDz );
	hipFree( d_corrDrDz );
	hipFree( d_corrDPhiRDz );
	hipFree( d_corrDz );
	hipFree( d_GDistDrDz );
	hipFree( d_GDistDPhiRDz );
	hipFree( d_GDistDz );
	hipFree( d_GCorrDrDz );
	hipFree( d_GCorrDPhiRDz );
	hipFree( d_GCorrDz );
	hipFree( d_rList );
	hipFree( d_zList );
	hipFree( d_phiList );
	hipFree( d_secondDerZDistDr);
	hipFree( d_secondDerZDistDPhiR);
	hipFree( d_secondDerZDistDz);
	hipFree( d_secondDerZCorrDr);
	hipFree( d_secondDerZCorrDPhiR);
	hipFree( d_secondDerZCorrDz);

	error = hipGetLastError();	
	if ( error != hipSuccess )
	{
		std::cout << "CUDA free allocated memory error: " << hipGetErrorString(error) << '\n';
	}
}






