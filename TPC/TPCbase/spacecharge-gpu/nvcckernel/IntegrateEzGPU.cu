#include "hip/hip_runtime.h"
#include "IntegrateEzGPU.h"

#include <hip/hip_runtime.h>
#include <math.h>

__device__ __constant__ float d_gridSizeZ;
__device__ __constant__ float d_ezField;
__device__ __constant__ int d_scanSize;
__device__ __constant__ int d_nRRow;
__device__ __constant__ int d_nZColumn;
__device__ __constant__ int d_phiSlice;
__device__ __constant__ int d_interpolationOrder;
__device__ __constant__ int d_currentZIndex;


__global__ void integrationCalculation
(
	float *d_arrayofIntEx,
	float *d_arrayofEx	
)
{
	extern __shared__ float temp[];

	int threadIndex = threadIdx.x;	
	int arrayIndex = blockIdx.x * (d_scanSize + 1);
	
	float first, second, last;
	
	int n = blockDim.x * 2;

	int offset = 1;
	
	// load data from input
	float temp_a = d_arrayofEx[arrayIndex + (2 * threadIndex)];
	float temp_b = d_arrayofEx[arrayIndex + (2 * threadIndex + 1)];

	// load last element from array to first variable
	first = d_arrayofEx[arrayIndex + d_scanSize];
	second = d_arrayofEx[arrayIndex + d_scanSize - 1];

/* odd function */
	// save data to shared memory flipped
	temp[(d_scanSize - 1) - (2 * threadIndex)] = 4 * temp_a;
	temp[(d_scanSize - 1) - (2 * threadIndex + 1)] = 2 * temp_b;
	
	// scan the array
	for (int d = n >> 1; d > 0; d >>= 1)
	{
		__syncthreads();
		
		if (threadIndex < d)
		{
			int ai = offset * (2 * threadIndex + 1) - 1;
			int bi = offset * (2 * threadIndex + 2) - 1;

			temp[bi] += temp[ai];
		}
		offset *= 2;
	}

	if (threadIndex == 0)
	{
		temp[n - 1] = 0;
	}

	for (int d = 1; d < n; d *= 2)
	{
		offset >>= 1;
		__syncthreads();

		if (threadIndex < d)
		{
			int ai = offset * (2 * threadIndex + 1) - 1;
			int bi = offset * (2 * threadIndex + 2) - 1;

			float t = temp[ai];
			temp[ai] = temp[bi];
			temp[bi] += t;
		}
	}
	__syncthreads();
	
	// save odd-numbered scan to even-numbered array
	d_arrayofIntEx[arrayIndex + (2 * threadIndex + 1)] = ((1.5 * first) + (0.5 * second) + temp[(d_scanSize - 1) - (2 * threadIndex)] - temp_b) * (d_gridSizeZ / 3.0) / (-1 * d_ezField);

/* even function */
	// save data to shared memory flipped
	temp[(d_scanSize - 1) - (2 * threadIndex)] = 2 * temp_a;
	temp[(d_scanSize - 1) - (2 * threadIndex + 1)] = 4 * temp_b;
	
	// scan the array
	for (int d = n >> 1; d > 0; d >>= 1)
	{
		__syncthreads();
		
		if (threadIndex < d)
		{
			int ai = offset * (2 * threadIndex + 1) - 1;
			int bi = offset * (2 * threadIndex + 2) - 1;

			temp[bi] += temp[ai];
		}
		offset *= 2;
	}

	if (threadIndex == 0)
	{
		last = temp[n - 1];		
		temp[n - 1] = 0;
	}

	for (int d = 1; d < n; d *= 2)
	{
		offset >>= 1;
		__syncthreads();

		if (threadIndex < d)
		{
			int ai = offset * (2 * threadIndex + 1) - 1;
			int bi = offset * (2 * threadIndex + 2) - 1;

			float t = temp[ai];
			temp[ai] = temp[bi];
			temp[bi] += t;
		}
	}
	__syncthreads();

	if (threadIndex == 0)
	{
		d_arrayofIntEx[arrayIndex + d_scanSize] = 0.0;
		d_arrayofIntEx[arrayIndex] = (first + last - temp_a) * (d_gridSizeZ / 3.0) / (-1 * d_ezField);
	}
	else
	{
		d_arrayofIntEx[arrayIndex + (2 * threadIndex)] = (first + temp[(d_scanSize - 1) - (2 * threadIndex) + 1] - temp_a) * (d_gridSizeZ / 3.0) / (-1 * d_ezField);
	}
}

extern "C" void IntegrateEzGPU 
(
	float *arrayOfIntEx, 
	float *arrayOfEx, 
	const int rows, 
	const int columns,  
	const int phislices, 
	float gridSizeZ, 
	float ezField	
)
{
	// initialize device array
	float *d_arrayofIntEx;
	float *d_arrayofEx;

	// set scan size to columns - 1
	int scanSize = columns - 1;

	std::cout << scanSize << std::endl;

	// set grid size and block size
	dim3 gridSize(rows * phislices);
	dim3 blockSize(scanSize / 2);

	// device memory allocation
	hipMalloc( &d_arrayofIntEx, rows * columns * phislices * sizeof(float) );
	hipMalloc( &d_arrayofEx, rows * columns * phislices * sizeof(float) );

	// copy data from host to device
	hipMemcpy( d_arrayofEx, arrayOfEx, rows * columns * phislices * sizeof(float), hipMemcpyHostToDevice );

	// copy constant to device memory
	hipMemcpyToSymbol(HIP_SYMBOL( d_gridSizeZ), &gridSizeZ, 1 * sizeof(float), 0, hipMemcpyHostToDevice );
	hipMemcpyToSymbol(HIP_SYMBOL( d_ezField), &ezField, 1 * sizeof(float), 0, hipMemcpyHostToDevice );
	hipMemcpyToSymbol(HIP_SYMBOL( d_scanSize), &scanSize, 1 * sizeof(int), 0, hipMemcpyHostToDevice );

	// run the kernel
	integrationCalculation<<< gridSize, blockSize, 2 * scanSize * sizeof(float) >>>( d_arrayofIntEx, d_arrayofEx );

	// copy result from device to host
	hipMemcpy( arrayOfIntEx, d_arrayofIntEx, rows * columns * phislices * sizeof(float), hipMemcpyDeviceToHost );

	// free device memory
	hipFree( d_arrayofIntEx );
	hipFree( d_arrayofEx );
}




__global__ void integrateDistEzDriftLineGPUKernel
(
	float *distDrDz,
	float *distDPhiRDz,
	float *distDz, 
	float *GDistDrDz,
	float *GDistDPhiRDz,
	float *GDistDz, 
	float *rList,
	float *zList,
	float *phiList,
	float *secondDerZDistDr,
	float *secondDerZDistDPhiR,
	float *secondDerZDistDz
)
{
	int index, index_x, index_y, index_z;
	
	// float gDistDrDz, gDistDPhiRDz, gDistDz;
	float lDistDrDz, lDistDPhiRDz, lDistDz;
	float currentPhi,currentRadius, currentZ;


	
	index = (blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

	index_x = index / (d_nZColumn * d_nRRow);
	
	if (index_x == 0)
	{
		index_y = index / d_nRRow;
	}
	else	
	{
		index_y = (index % (index_x * d_nRRow * d_nZColumn)) / d_nRRow;
	}

	index_z = index % d_nZColumn;
	


	index = index_x * d_nRRow * d_nZColumn + index_y * d_nZColumn + index_z;	



	

	
	if ((index_x >= 0) && (index_x < d_phiSlice) && (index_y >= 0) && (index_y < d_nRRow ) && (index_z >= 0) && (index_z < d_nZColumn - 1) && (index_z >= d_currentZIndex)) {
		lDistDrDz = 0.0;
		lDistDPhiRDz = 0.0;
		lDistDz = 0.0;
		
		if (index_z == d_currentZIndex) {
			GDistDrDz[index] == 0.0;
			GDistDPhiRDz[index] = 0.0;
			GDistDz[index] = 0.0;
		} 
		currentRadius = rList[index_y] + GDistDrDz[index];
		currentPhi = phiList[index_x] + (GDistDPhiRDz[index]/currentRadius);
		if (currentPhi < 0.0) currentPhi = 2 * M_PI + currentPhi;
		if (currentPhi > 2*M_PI) currentPhi = currentPhi - (2 * M_PI);
		currentZ =  zList[d_currentZIndex] + GDistDz[index];

		// get Local Distortion through interpolation
		
		// update global distortion
		GDistDrDz[index] += lDistDrDz;
		GDistDPhiRDz[index] += lDistDPhiRDz;
		GDistDz[index] += lDistDz;
		
			
	}

}



extern "C" void IntegrateEzDriftLineGPU(
	float * distDrDz, float * distDPhiRDz, float * distDz, float *corrDrDz, float * corrDPhiRDz, float * corrDz,  
	float * GDistDrDz, float * GDistDPhiRDz, float * GDistDz, float * GCorrDrDz, float * GCorrDPhiRDz, float * GCorrDz,  
	float * rList, float * zList, float * phiList,   
	const int rows, const int columns, const int phislices, const int interpolationOrder,
	float * secondDerZDistDr, float *secondDerZDistDPhiR, float *secondDerZDistDz,
	float * secondDerZCorrDr, float *secondDerZCorrDPhiR, float *secondDerZCorrDz) {



	// initialize device array
	float *d_distDrDz;
	float *d_distDPhiRDz;
	float *d_distDz;
	float *d_corrDrDz;
	float *d_corrDPhiRDz;
	float *d_corrDz;
	float *d_GDistDrDz;
	float *d_GDistDPhiRDz;
	float *d_GDistDz;
	float *d_GCorrDrDz;
	float *d_GCorrDPhiRDz;
	float *d_GCorrDz;
	float *d_rList;
	float *d_zList;
	float *d_phiList;
	float *d_secondDerZDistDr;
	float *d_secondDerZDistDPhiR;
	float *d_secondDerZDistDz;

	float *d_secondDerZCorrDr;
	float *d_secondDerZCorrDPhiR;
	float *d_secondDerZCorrDz;

	int *d_currentZIndex;
	int currentZIndex;

	hipError_t error;

	hipMalloc( &d_distDrDz, rows * columns * phislices * sizeof(float) );
	hipMalloc( &d_distDPhiRDz, rows * columns * phislices * sizeof(float) );
	hipMalloc( &d_distDz, rows * columns * phislices * sizeof(float) );
	hipMalloc( &d_corrDrDz, rows * columns * phislices * sizeof(float) );
	hipMalloc( &d_corrDPhiRDz, rows * columns * phislices * sizeof(float) );
	hipMalloc( &d_corrDz, rows * columns * phislices * sizeof(float) );
	hipMalloc( &d_GDistDrDz, rows * columns * phislices * sizeof(float) );
	hipMalloc( &d_GDistDPhiRDz, rows * columns * phislices * sizeof(float) );
	hipMalloc( &d_GDistDz, rows * columns * phislices * sizeof(float) );
	hipMalloc( &d_GCorrDrDz, rows * columns * phislices * sizeof(float) );
	hipMalloc( &d_GCorrDPhiRDz, rows * columns * phislices * sizeof(float) );
	hipMalloc( &d_GCorrDz, rows * columns * phislices * sizeof(float) );
	hipMalloc( &d_rList, rows *  sizeof(float) );
	hipMalloc( &d_zList, columns *  sizeof(float) );
	hipMalloc( &d_phiList,  phislices * sizeof(float) );

	hipMalloc( &d_secondDerZDistDr, rows *  columns * phislices *  sizeof(float) );
	hipMalloc( &d_secondDerZDistDPhiR, rows *  columns * phislices *  sizeof(float) );
	hipMalloc( &d_secondDerZDistDz, rows *  columns * phislices *  sizeof(float) );
	
	hipMalloc( &d_secondDerZCorrDr, rows *  columns * phislices *  sizeof(float) );
	hipMalloc( &d_secondDerZCorrDPhiR, rows *  columns * phislices *  sizeof(float) );
	hipMalloc( &d_secondDerZCorrDz, rows *  columns * phislices *  sizeof(float) );



	error = hipGetLastError();	
	if ( error != hipSuccess )
	{    	
		std::cout << "CUDA memory allocation error: " << hipGetErrorString(error) << '\n';
	}


	// copy from CPU to GPU
	// copy local distortion 
	hipMemcpy( d_distDrDz, distDrDz, rows * columns * phislices * sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( d_distDPhiRDz, distDPhiRDz, rows * columns * phislices * sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( d_distDz, distDz, rows * columns * phislices * sizeof(float), hipMemcpyHostToDevice );

	hipMemcpy( d_corrDrDz, corrDrDz, rows * columns * phislices * sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( d_corrDPhiRDz, corrDPhiRDz, rows * columns * phislices * sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( d_corrDz, corrDz, rows * columns * phislices * sizeof(float), hipMemcpyHostToDevice );

	hipMemcpy( d_rList, rList, rows  * sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( d_zList, zList, columns *  sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( d_phiList, phiList,  phislices * sizeof(float), hipMemcpyHostToDevice );
	
	hipMemcpy( d_secondDerZDistDr, secondDerZDistDr, rows * columns * phislices * sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( d_secondDerZDistDPhiR, secondDerZDistDPhiR, rows * columns * phislices * sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( d_secondDerZDistDz, secondDerZDistDz, rows * columns * phislices * sizeof(float), hipMemcpyHostToDevice );

	hipMemcpy( d_secondDerZCorrDr, secondDerZCorrDr, rows * columns * phislices * sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( d_secondDerZCorrDPhiR, secondDerZCorrDPhiR, rows * columns * phislices * sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( d_secondDerZCorrDz, secondDerZCorrDz, rows * columns * phislices * sizeof(float), hipMemcpyHostToDevice );
	hipMemcpyToSymbol(HIP_SYMBOL( d_interpolationOrder), &interpolationOrder, 1 * sizeof(int), 0, hipMemcpyHostToDevice );
	hipMemcpyToSymbol(HIP_SYMBOL( d_nRRow), &rows, 1 * sizeof(int), 0, hipMemcpyHostToDevice );
	hipMemcpyToSymbol(HIP_SYMBOL( d_nZColumn), &columns, 1 * sizeof(int), 0, hipMemcpyHostToDevice );
	hipMemcpyToSymbol(HIP_SYMBOL( d_phiSlice), &phislices, 1 * sizeof(int), 0, hipMemcpyHostToDevice );
	error = hipGetLastError();	
	if ( error != hipSuccess )
	{
		std::cout << "CUDA memory copy host to device error: " << hipGetErrorString(error) << '\n';
	}

	// call kernel
	// set grid size and block size
	dim3 gridSize((rows / 32) + 1, (columns / 32) + 1, phislices);
	dim3 blockSize(32, 32);

	for (currentZIndex = 0; currentZIndex < columns -1;currentZIndex++) {	
		hipMemcpyToSymbol(HIP_SYMBOL(d_currentZIndex),&currentZIndex, 1 * sizeof(int), 0, hipMemcpyHostToDevice);

		integrateDistEzDriftLineGPUKernel<<< gridSize,blockSize >>>(d_distDrDz,d_distDPhiRDz,d_distDz,
					  d_GDistDrDz,d_GDistDPhiRDz,d_GDistDz, 
					  d_rList,d_zList, d_phiList,
					  d_secondDerZDistDr, d_secondDerZDistDPhiR, d_secondDerZDistDz);
	}
	
	hipMemcpy( GDistDrDz, d_GDistDrDz, rows * columns * phislices * sizeof(float), hipMemcpyDeviceToHost );
	hipMemcpy( GDistDPhiRDz, d_GDistDPhiRDz, rows * columns * phislices * sizeof(float), hipMemcpyDeviceToHost );
	hipMemcpy( GDistDz, d_GDistDz, rows * columns * phislices * sizeof(float), hipMemcpyDeviceToHost );
	hipMemcpy( GCorrDrDz, d_GCorrDrDz, rows * columns * phislices * sizeof(float), hipMemcpyDeviceToHost );
	hipMemcpy( GCorrDPhiRDz, d_GCorrDPhiRDz, rows * columns * phislices * sizeof(float), hipMemcpyDeviceToHost );
	hipMemcpy( GCorrDz, d_GCorrDz, rows * columns * phislices * sizeof(float), hipMemcpyDeviceToHost );
	error = hipGetLastError();	
	if ( error != hipSuccess )
	{
		std::cout << "CUDA memory copy device to host error: " << hipGetErrorString(error) << '\n';
	}

	hipFree( d_distDrDz );
	hipFree( d_distDPhiRDz );
	hipFree( d_distDz );
	hipFree( d_corrDrDz );
	hipFree( d_corrDPhiRDz );
	hipFree( d_corrDz );
	hipFree( d_GDistDrDz );
	hipFree( d_GDistDPhiRDz );
	hipFree( d_GDistDz );
	hipFree( d_GCorrDrDz );
	hipFree( d_GCorrDPhiRDz );
	hipFree( d_GCorrDz );
	hipFree( d_rList );
	hipFree( d_zList );
	hipFree( d_phiList );
	hipFree( d_secondDerZDistDr);
	hipFree( d_secondDerZDistDPhiR);
	hipFree( d_secondDerZDistDz);
	hipFree( d_secondDerZCorrDr);
	hipFree( d_secondDerZCorrDPhiR);
	hipFree( d_secondDerZCorrDz);

	error = hipGetLastError();	
	if ( error != hipSuccess )
	{
		std::cout << "CUDA free allocated memory error: " << hipGetErrorString(error) << '\n';
	}
}






