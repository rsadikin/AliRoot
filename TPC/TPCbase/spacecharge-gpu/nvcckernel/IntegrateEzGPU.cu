#include "hip/hip_runtime.h"
#include "IntegrateEzGPU.h"

#include <hip/hip_runtime.h>

__device__ __constant__ float d_gridSizeZ;
__device__ __constant__ float d_ezField;
__device__ __constant__ int d_scanSize;

__global__ void integrationCalculation
(
	float *d_arrayofIntEx,
	float *d_arrayofEx	
)
{
	extern __shared__ float temp[];

	int threadIndex = threadIdx.x;	
	int arrayIndex = blockIdx.x * (d_scanSize + 1);
	
	float first, second, last;
	
	int n = blockDim.x * 2;

	int offset = 1;
	
	// load data from input
	float temp_a = d_arrayofEx[arrayIndex + (2 * threadIndex)];
	float temp_b = d_arrayofEx[arrayIndex + (2 * threadIndex + 1)];

	// load last element from array to first variable
	first = d_arrayofEx[arrayIndex + d_scanSize];
	second = d_arrayofEx[arrayIndex + d_scanSize - 1];

/* odd function */
	// save data to shared memory flipped
	temp[(d_scanSize - 1) - (2 * threadIndex)] = 4 * temp_a;
	temp[(d_scanSize - 1) - (2 * threadIndex + 1)] = 2 * temp_b;
	
	// scan the array
	for (int d = n >> 1; d > 0; d >>= 1)
	{
		__syncthreads();
		
		if (threadIndex < d)
		{
			int ai = offset * (2 * threadIndex + 1) - 1;
			int bi = offset * (2 * threadIndex + 2) - 1;

			temp[bi] += temp[ai];
		}
		offset *= 2;
	}

	if (threadIndex == 0)
	{
		temp[n - 1] = 0;
	}

	for (int d = 1; d < n; d *= 2)
	{
		offset >>= 1;
		__syncthreads();

		if (threadIndex < d)
		{
			int ai = offset * (2 * threadIndex + 1) - 1;
			int bi = offset * (2 * threadIndex + 2) - 1;

			float t = temp[ai];
			temp[ai] = temp[bi];
			temp[bi] += t;
		}
	}
	__syncthreads();
	
	// save odd-numbered scan to even-numbered array
	d_arrayofIntEx[arrayIndex + (2 * threadIndex + 1)] = ((1.5 * first) + (0.5 * second) + temp[(d_scanSize - 1) - (2 * threadIndex)] - temp_b) * (d_gridSizeZ / 3.0) / (-1 * d_ezField);

/* even function */
	// save data to shared memory flipped
	temp[(d_scanSize - 1) - (2 * threadIndex)] = 2 * temp_a;
	temp[(d_scanSize - 1) - (2 * threadIndex + 1)] = 4 * temp_b;
	
	// scan the array
	for (int d = n >> 1; d > 0; d >>= 1)
	{
		__syncthreads();
		
		if (threadIndex < d)
		{
			int ai = offset * (2 * threadIndex + 1) - 1;
			int bi = offset * (2 * threadIndex + 2) - 1;

			temp[bi] += temp[ai];
		}
		offset *= 2;
	}

	if (threadIndex == 0)
	{
		last = temp[n - 1];		
		temp[n - 1] = 0;
	}

	for (int d = 1; d < n; d *= 2)
	{
		offset >>= 1;
		__syncthreads();

		if (threadIndex < d)
		{
			int ai = offset * (2 * threadIndex + 1) - 1;
			int bi = offset * (2 * threadIndex + 2) - 1;

			float t = temp[ai];
			temp[ai] = temp[bi];
			temp[bi] += t;
		}
	}
	__syncthreads();

	if (threadIndex == 0)
	{
		d_arrayofIntEx[arrayIndex + d_scanSize] = 0.0;
		d_arrayofIntEx[arrayIndex] = (first + last - temp_a) * (d_gridSizeZ / 3.0) / (-1 * d_ezField);
	}
	else
	{
		d_arrayofIntEx[arrayIndex + (2 * threadIndex)] = (first + temp[(d_scanSize - 1) - (2 * threadIndex) + 1] - temp_a) * (d_gridSizeZ / 3.0) / (-1 * d_ezField);
	}
}

extern "C" void IntegrateEzGPU 
(
	float *arrayOfIntEx, 
	float *arrayOfEx, 
	const int rows, 
	const int columns,  
	const int phislices, 
	float gridSizeZ, 
	float ezField	
)
{
	// initialize device array
	float *d_arrayofIntEx;
	float *d_arrayofEx;

	// set scan size to columns - 1
	int scanSize = columns - 1;

	std::cout << scanSize << std::endl;

	// set grid size and block size
	dim3 gridSize(rows * phislices);
	dim3 blockSize(scanSize / 2);

	// device memory allocation
	hipMalloc( &d_arrayofIntEx, rows * columns * phislices * sizeof(float) );
	hipMalloc( &d_arrayofEx, rows * columns * phislices * sizeof(float) );

	// copy data from host to device
	hipMemcpy( d_arrayofEx, arrayOfEx, rows * columns * phislices * sizeof(float), hipMemcpyHostToDevice );

	// copy constant to device memory
	hipMemcpyToSymbol(HIP_SYMBOL( d_gridSizeZ), &gridSizeZ, 1 * sizeof(float), 0, hipMemcpyHostToDevice );
	hipMemcpyToSymbol(HIP_SYMBOL( d_ezField), &ezField, 1 * sizeof(float), 0, hipMemcpyHostToDevice );
	hipMemcpyToSymbol(HIP_SYMBOL( d_scanSize), &scanSize, 1 * sizeof(int), 0, hipMemcpyHostToDevice );

	// run the kernel
	integrationCalculation<<< gridSize, blockSize, 2 * scanSize * sizeof(float) >>>( d_arrayofIntEx, d_arrayofEx );

	// copy result from device to host
	hipMemcpy( arrayOfIntEx, d_arrayofIntEx, rows * columns * phislices * sizeof(float), hipMemcpyDeviceToHost );

	// free device memory
	hipFree( d_arrayofIntEx );
	hipFree( d_arrayofEx );
}



__device__ void interpolateGPUKernel
(
	float *rLookUp,
	float *zLookUp,
	float *phiRLookUp,
	float *rList,
	float *zList,
	float *phiList,
	float r,
	float z,
	float phi,
	float *valueR,
	float *valueZ,
	float *valuePhi
)
{
}

__global__ void integrateEzDriftLineGPUKernel
(
	float *distDrDz,
	float *distDPhiRDz,
	float *distDz, 
	float *corrDrDz,
	float *corrDPhiRDz,
	float *corrDz, 
	float *GDistDrDz,
	float *GDistDPhiRDz,
	float *GDistDz, 
	float *GCorrDrDz,
	float *GCorrDPhiRDz,
	float *GCorrDz,
	float *rList,
	float *zList,
	float *phiList, 
	const int rows,
	const int columns,
	const int phislices
)
{
	int index, index_x, index_y, index_z;
	
	
	float gDistDrDz, gDistDPhiRDz, gDistDz;
	float lDistDrDz, lDistDPhiRDz, lDistDz;
	float phi,radius, z;	
	index = (blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

	index_x = index / (rows * columns);
	
	if (index_x == 0)
	{
		index_y = index / rows;
	}
	else	
	{
		index_y = (index % (index_x * rows * columns)) / rows;
	}

	index_z = index % columns;
	


	index = index_x * rows * columns + index_y * columns + index_z;	

	// total distortion
	gDistDrDz = 0.0;
	gDistDPhiRDz = 0.0;
	gDistDz = 0.0;

	// starting point
	phi = phiList[index_x];
	z   = zList[index_z];
	radius = rList[index_y];

	for (int jj=index_z; jj < columns;jj++) {
		// interpolate local distortion 
		interpolateGPUKernel(distDrDz,distDz,distDPhiRDz,rList,zList,phiList,radius,z,phi,&lDistDrDz,&lDistDz,&lDistDPhiRDz);
		
	}


}




extern "C" void IntegrateEzDriftLineGPU(float * distDrDz, float * distDPhiRDz, float * distDz, float *corrDrDz, float * corrDPhiRDz, float * corrDz,  
	float * GDistDrDz, float * GDistDPhiRDz, float * GDistDz, float * GCorrDrDz, float * GCorrDPhiRDz, float * GCorrDz,  
	float * rList, float * zList, float * phiList,   
	const int rows, const int columns, const int phislices, const int integrationType) {

	// initialize device array
	float *d_distDrDz;
	float *d_distDPhiRDz;
	float *d_distDz;
	float *d_corrDrDz;
	float *d_corrDPhiRDz;
	float *d_corrDz;
	float *d_GDistDrDz;
	float *d_GDistDPhiRDz;
	float *d_GDistDz;
	float *d_GCorrDrDz;
	float *d_GCorrDPhiRDz;
	float *d_GCorrDz;
	float *d_rList;
	float *d_zList;
	float *d_phiList;
	
	hipError_t error;

	hipMalloc( &d_distDrDz, rows * columns * phislices * sizeof(float) );
	hipMalloc( &d_distDPhiRDz, rows * columns * phislices * sizeof(float) );
	hipMalloc( &d_distDz, rows * columns * phislices * sizeof(float) );
	hipMalloc( &d_corrDrDz, rows * columns * phislices * sizeof(float) );
	hipMalloc( &d_corrDPhiRDz, rows * columns * phislices * sizeof(float) );
	hipMalloc( &d_corrDz, rows * columns * phislices * sizeof(float) );
	hipMalloc( &d_GDistDrDz, rows * columns * phislices * sizeof(float) );
	hipMalloc( &d_GDistDPhiRDz, rows * columns * phislices * sizeof(float) );
	hipMalloc( &d_GDistDz, rows * columns * phislices * sizeof(float) );
	hipMalloc( &d_GCorrDrDz, rows * columns * phislices * sizeof(float) );
	hipMalloc( &d_GCorrDPhiRDz, rows * columns * phislices * sizeof(float) );
	hipMalloc( &d_GCorrDz, rows * columns * phislices * sizeof(float) );
	hipMalloc( &d_rList, rows *  sizeof(float) );
	hipMalloc( &d_zList, columns *  sizeof(float) );
	hipMalloc( &d_phiList,  phislices * sizeof(float) );

	error = hipGetLastError();	
	if ( error != hipSuccess )
	{    	
		std::cout << "CUDA memory allocation error: " << hipGetErrorString(error) << '\n';
	}


	// copy from CPU to GPU
	// copy local distortion 
	hipMemcpy( d_distDrDz, distDrDz, rows * columns * phislices * sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( d_distDPhiRDz, distDPhiRDz, rows * columns * phislices * sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( d_distDz, distDz, rows * columns * phislices * sizeof(float), hipMemcpyHostToDevice );

	hipMemcpy( d_corrDrDz, corrDrDz, rows * columns * phislices * sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( d_corrDPhiRDz, corrDPhiRDz, rows * columns * phislices * sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( d_corrDz, corrDz, rows * columns * phislices * sizeof(float), hipMemcpyHostToDevice );

	hipMemcpy( d_rList, rList, rows  * sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( d_zList, zList, columns *  sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( d_phiList, phiList,  phislices * sizeof(float), hipMemcpyHostToDevice );
	error = hipGetLastError();	
	if ( error != hipSuccess )
	{
		std::cout << "CUDA memory copy host to device error: " << hipGetErrorString(error) << '\n';
	}

	// call kernel
	// set grid size and block size
	dim3 gridSize((rows / 32) + 1, (columns / 32) + 1, phislices);
	dim3 blockSize(32, 32);


	
	hipMemcpy( GDistDrDz, d_GDistDrDz, rows * columns * phislices * sizeof(float), hipMemcpyDeviceToHost );
	hipMemcpy( GDistDPhiRDz, d_GDistDPhiRDz, rows * columns * phislices * sizeof(float), hipMemcpyDeviceToHost );
	hipMemcpy( GDistDz, d_GDistDz, rows * columns * phislices * sizeof(float), hipMemcpyDeviceToHost );
	hipMemcpy( GCorrDrDz, d_GCorrDrDz, rows * columns * phislices * sizeof(float), hipMemcpyDeviceToHost );
	hipMemcpy( GCorrDPhiRDz, d_GCorrDPhiRDz, rows * columns * phislices * sizeof(float), hipMemcpyDeviceToHost );
	hipMemcpy( GCorrDz, d_GCorrDz, rows * columns * phislices * sizeof(float), hipMemcpyDeviceToHost );
	error = hipGetLastError();	
	if ( error != hipSuccess )
	{
		std::cout << "CUDA memory copy device to host error: " << hipGetErrorString(error) << '\n';
	}

	hipFree( d_distDrDz );
	hipFree( d_distDPhiRDz );
	hipFree( d_distDz );
	hipFree( d_corrDrDz );
	hipFree( d_corrDPhiRDz );
	hipFree( d_corrDz );
	hipFree( d_GDistDrDz );
	hipFree( d_GDistDPhiRDz );
	hipFree( d_GDistDz );
	hipFree( d_GCorrDrDz );
	hipFree( d_GCorrDPhiRDz );
	hipFree( d_GCorrDz );
	hipFree( d_rList );
	hipFree( d_zList );
	hipFree( d_phiList );

	error = hipGetLastError();	
	if ( error != hipSuccess )
	{
		std::cout << "CUDA free allocated memory error: " << hipGetErrorString(error) << '\n';
	}
}






