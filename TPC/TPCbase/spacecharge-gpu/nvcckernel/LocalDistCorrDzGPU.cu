#include "hip/hip_runtime.h"
#include "LocalDistCorrDzGPU.h"
#include <hip/hip_runtime.h>
#include <math.h>

__device__ __constant__ float d_gridSizeZ;
__device__ __constant__ float d_ezField;
__device__ __constant__ float d_fC0;
__device__ __constant__ float d_fC1;
__device__ __constant__ float d_fgkdvdE;

__global__ void localDistCorrDzGPUKernel
(
	float *matEr,
	float *matEz,
	float *matEPhi, 
	float *matDistDrDz,
	float *matDistDPhiRDz,
	float *matDistDz, 
	float *matCorrDrDz,
	float *matCorrDPhiRDz,
	float *matCorrDz, 
	const int rows,
	const int columns,
	const int phislices
)
{
	int index, index_x, index_y, index_z;

	
	float localIntErOverEz, localIntEPhiOverEz, localIntDeltaEz;	
	index = (blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

	index_x = index / (rows * columns);
	
	if (index_x == 0)
	{
		index_y = index / rows;
	}
	else	
	{
		index_y = (index % (index_x * rows * columns)) / rows;
	}

	index_z = index % columns;
	
	
	if ((index_x >= 0) && (index_x < phislices) && (index_y > 0) && (index_y < rows - 1) && (index_z > 0) && (index_z < columns - 1))
	{
        	localIntErOverEz = (d_gridSizeZ / 2.0) * (matEr[index_x * rows * columns + index_y * columns + index_z ] + matEr[index_x * rows * columns + index_y * columns + index_z + 1]) / (-1 * d_ezField);
        	localIntEPhiOverEz = (d_gridSizeZ / 2.0) * (matEPhi[index_x * rows * columns + index_y * columns + index_z ] + matEPhi[index_x * rows * columns + index_y * columns + index_z + 1]) / (-1 * d_ezField);
        	localIntDeltaEz = (d_gridSizeZ / 2.0) * (matEz[index_x * rows * columns + index_y * columns + index_z ] + matEz[index_x * rows * columns + index_y * columns + index_z + 1]) ;


		matDistDrDz[index_x * rows  *columns + index_y *columns + index_z] = d_fC0 * localIntErOverEz + d_fC1 * localIntEPhiOverEz;
		matDistDPhiRDz[index_x * rows  *columns + index_y *columns + index_z] = d_fC0 * localIntEPhiOverEz - d_fC1 * localIntErOverEz;
		matDistDz[index_x * rows  *columns + index_y *columns + index_z] = d_fgkdvdE * d_fgkdvdE * localIntDeltaEz;

		matCorrDrDz[index_x * rows  *columns + index_y *columns + index_z + 1] = -1 * matDistDrDz[index_x * rows  *columns + index_y *columns + index_z ]; 
		matCorrDPhiRDz[index_x * rows  *columns + index_y *columns + index_z + 1] = -1 * matDistDPhiRDz[index_x * rows  *columns + index_y *columns + index_z ]; 
		matCorrDz[index_x * rows  *columns + index_y *columns + index_z + 1] = -1 * matDistDz[index_x * rows  *columns + index_y *columns + index_z ]; 


	}

}





extern "C" void LocalDistCorrDzGPU (
	float *matEr,
	float *matEz,
	float *matEPhi, 
	float *matDistDrDz,
	float *matDistDPhiRDz,
	float *matDistDz, 
	float *matCorrDrDz,
	float *matCorrDPhiRDz,
	float *matCorrDz, 
	const int rows,
	const int columns,
	const int phislices,
	const float gridSizeZ,
	const float ezField,	
  	const float fC0,
	const float fC1,
	const float fgkdvdE	
)
{
	// device array
	float *d_matEr;
	float *d_matEz;
	float *d_matEPhi;
	float *d_matDistDrDz;
	float *d_matDistDPhiRDz;
	float *d_matDistDz;
	float *d_matCorrDrDz;
	float *d_matCorrDPhiRDz;
	float *d_matCorrDz;

	hipError_t error;

	// pre-compute constant

	
	// device memory allocation
	hipMalloc( &d_matEr, rows * columns * phislices * sizeof(float) );
	hipMalloc( &d_matEz, rows * columns * phislices * sizeof(float) );
	hipMalloc( &d_matEPhi, rows * columns * phislices * sizeof(float) );
	hipMalloc( &d_matDistDrDz, rows * columns * phislices * sizeof(float) );
	hipMalloc( &d_matDistDPhiRDz, rows * columns * phislices * sizeof(float) );
	hipMalloc( &d_matDistDz, rows * columns * phislices * sizeof(float) );
	hipMalloc( &d_matCorrDrDz, rows * columns * phislices * sizeof(float) );
	hipMalloc( &d_matCorrDPhiRDz, rows * columns * phislices * sizeof(float) );
	hipMalloc( &d_matCorrDz, rows * columns * phislices * sizeof(float) );

	error = hipGetLastError();	
	if ( error != hipSuccess )
	{    	
		std::cout << "CUDA memory allocation error: " << hipGetErrorString(error) << '\n';
	}

	// copy data from host to device
	hipMemcpy( d_matEr, matEr, rows * columns * phislices * sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( d_matEPhi, matEPhi, rows * columns * phislices * sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( d_matEz, matEz, rows * columns * phislices * sizeof(float), hipMemcpyHostToDevice );

	error = hipGetLastError();	
	if ( error != hipSuccess )
	{
		std::cout << "CUDA memory copy host to device error: " << hipGetErrorString(error) << '\n';
	}

	// copy constant from host to device
	hipMemcpyToSymbol(HIP_SYMBOL( d_gridSizeZ), &gridSizeZ, 1 * sizeof(float), 0, hipMemcpyHostToDevice );
	hipMemcpyToSymbol(HIP_SYMBOL( d_ezField), &ezField, 1 * sizeof(float), 0, hipMemcpyHostToDevice );
	hipMemcpyToSymbol(HIP_SYMBOL( d_fC0), &fC0, 1 * sizeof(float), 0, hipMemcpyHostToDevice );
	hipMemcpyToSymbol(HIP_SYMBOL( d_fC1), &fC1, 1 * sizeof(float), 0, hipMemcpyHostToDevice );
	hipMemcpyToSymbol(HIP_SYMBOL( d_fgkdvdE), &fgkdvdE, 1 * sizeof(float), 0, hipMemcpyHostToDevice );

	error = hipGetLastError();	
	if ( error != hipSuccess )
	{
		std::cout << "CUDA memory copy to constant memory host to device error: " << hipGetErrorString(error) << '\n';
	}

	// set grid size and block size
	dim3 gridSize((rows / 32) + 1, (columns / 32) + 1, phislices);
	dim3 blockSize(32, 32);

	// run the kernel
 	localDistCorrDzGPUKernel<<< gridSize, blockSize >>>( d_matEr, d_matEz, d_matEPhi, d_matDistDrDz, d_matDistDPhiRDz, d_matDistDz, d_matCorrDrDz, d_matCorrDPhiRDz, d_matCorrDz, rows, columns, phislices );

	error = hipGetLastError();	
	if ( error != hipSuccess )
	{
		std::cout << "CUDA kernel run error: " << hipGetErrorString(error) << '\n';
	}

	// copy result from device to host
	hipMemcpy( matDistDrDz, d_matDistDrDz, rows * columns * phislices * sizeof(float), hipMemcpyDeviceToHost );
	hipMemcpy( matDistDPhiRDz, d_matDistDPhiRDz, rows * columns * phislices * sizeof(float), hipMemcpyDeviceToHost );
	hipMemcpy( matDistDz, d_matDistDz, rows * columns * phislices * sizeof(float), hipMemcpyDeviceToHost );
	hipMemcpy( matCorrDrDz, d_matCorrDrDz, rows * columns * phislices * sizeof(float), hipMemcpyDeviceToHost );
	hipMemcpy( matCorrDPhiRDz, d_matCorrDPhiRDz, rows * columns * phislices * sizeof(float), hipMemcpyDeviceToHost );
	hipMemcpy( matCorrDz, d_matCorrDz, rows * columns * phislices * sizeof(float), hipMemcpyDeviceToHost );

	error = hipGetLastError();	
	if ( error != hipSuccess )
	{
		std::cout << "CUDA memory copy device to host error: " << hipGetErrorString(error) << '\n';
	}

	// free device memory
	hipFree( matEr );
	hipFree( matEPhi );
	hipFree( matEz );
	
	hipFree( matDistDrDz );
	hipFree( matDistDPhiRDz );
	hipFree( matDistDz );
	hipFree( matCorrDrDz );
	hipFree( matCorrDPhiRDz );
	hipFree( matCorrDz );

	error = hipGetLastError();	
	if ( error != hipSuccess )
	{
		std::cout << "CUDA free allocated memory error: " << hipGetErrorString(error) << '\n';
	}
}

